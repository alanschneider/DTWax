#ifndef NORMALIZER
#define NORMALIZER

#include "common.hpp"
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDNN_ASSERT(func)                                                     \
  {                                                                            \
    hipdnnStatus_t e = (func);                                                  \
    std::cout << "\ncuDTW::  cuDNN Normalizer returned: "                      \
              << hipdnnGetErrorString(e) << "\n";                               \
  }

// normalizer class
class normalizer {
public:
  void normalize(raw_t *raw_squiggle_array, index_t num_reads, index_t length);
  normalizer(); // CUDNN normalizer
  ~normalizer();
  void print_normalized_query(raw_t *raw_array, index_t NUM_READS,
                              std::vector<std::string> &read_ids);

private:
  float *bnScale, *bnBias;
  float bnScale_h[QUERY_LEN], bnBias_h[QUERY_LEN];
  float alpha[1] = {1};
  float beta[1] = {0.0};
};

void normalizer::print_normalized_query(raw_t *raw_array, index_t NUM_READS,
                                        std::vector<std::string> &read_ids) {
  std::cout << "Normalized query:\n";
  for (index_t i = 0; i < NUM_READS; i++) {
    std::cout << "cuDTW:: " << read_ids[i] << "\n";
    for (index_t j = 0; j < QUERY_LEN; j++) {
      std::cout << raw_array[(i * QUERY_LEN + j)] << ",";
    }
    std::cout << "\n";
  }
  std::cout << "\n=================\n";
}
normalizer::~normalizer() {
  hipFree(bnScale);
  hipFree(bnBias);
}
normalizer::normalizer(void) {

  // create scale and bias vectors
  for (int i = 0; i < QUERY_LEN; i++) {
    bnScale_h[i] = 1.0f;
    bnBias_h[i] = 0.0f;
  }
  hipMalloc(&bnScale, (QUERY_LEN * sizeof(float)));
  hipMalloc(&bnBias, (QUERY_LEN * sizeof(float)));

  hipMemcpyAsync(bnScale,
                  &bnScale_h[0], //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!//
                  sizeof(float) * QUERY_LEN, hipMemcpyHostToDevice);
  hipMemcpyAsync(bnBias,
                  &bnBias_h[0], //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!//
                  sizeof(float) * QUERY_LEN, hipMemcpyHostToDevice);
}

__inline__ void normalizer::normalize(raw_t *raw_squiggle_array,
                                      index_t num_reads, index_t length) {

  int c = num_reads, h = length; // nchw format for cudnn

  raw_t *x; // output,input array

  hipMalloc(&x, (sizeof(raw_t) * c * h));

  hipMemcpy(x, &raw_squiggle_array[0], (sizeof(raw_t) * c * h),
             hipMemcpyHostToDevice);

  hipdnnHandle_t handle_;
  hipdnnCreate(&handle_);
  hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
  hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW; // HIPDNN_TENSOR_NCHW;
  hipdnnBatchNormMode_t mode = HIPDNN_BATCHNORM_SPATIAL;

  // descriptors
  hipdnnTensorDescriptor_t x_desc, bnScaleBiasMeanVarDesc;
  hipdnnCreateTensorDescriptor(&x_desc);
  hipdnnSetTensor4dDescriptor(x_desc, format, dtype, 1, c, h, 1);

  hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc);
  hipdnnDeriveBNTensorDescriptor(bnScaleBiasMeanVarDesc, x_desc, mode);

  // normalize
  CUDNN_ASSERT(hipdnnBatchNormalizationForwardTraining(
      handle_, mode, alpha, beta, x_desc, x, x_desc, x, bnScaleBiasMeanVarDesc,
      bnScale, bnBias, 1.0 / (1.0 + h), NULL, NULL, 0.0001f, NULL, NULL));

  hipMemcpy(

      &raw_squiggle_array[0], x, (sizeof(raw_t) * c * h),
      hipMemcpyDeviceToHost);

  // std::cout << "cudnn normalized output:\n";
  // for (uint64_t i = 0; i < (uint64_t)((int64_t)QUERY_LEN *
  // (int64_t)NUM_READS);
  //      i++) {

  //   std::cout << raw_squiggle_array[i] << ",";
  // }
  hipdnnDestroy(handle_);
  return;
}

#endif
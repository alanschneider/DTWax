#include <assert.h>
#include <cstdint>
#include <iostream>
#include <string>

#include "include/DTW.hpp"
#include "include/binary_IO.hpp"
#include "include/cbf_generator.hpp"
#include "include/common.hpp"
#include "include/hpc_helpers.hpp"

using namespace FullDTW;

//------------------time macros--------------------------//
#define TIMERSTART_CUDA(label)                                                 \
  hipSetDevice(0);                                                            \
  hipEvent_t start##label, stop##label;                                       \
  float time##label;                                                           \
  hipEventCreate(&start##label);                                              \
  hipEventCreate(&stop##label);                                               \
  hipEventRecord(start##label, 0);

#define TIMERSTOP_CUDA(label)                                                  \
  hipSetDevice(0);                                                            \
  hipEventRecord(stop##label, 0);                                             \
  hipEventSynchronize(stop##label);                                           \
  hipEventElapsedTime(&time##label, start##label, stop##label);               \
  std::cout << "TIMING: " << time##label << " ms "                             \
            << ((num_features + 1) * (num_features + 1) * num_entries *        \
                num_entries) /                                                 \
                   (time##label * 1e6)                                         \
            << " GCUPS (" << #label << ")" << std::endl;
//..................time macros............................//

int main(int argc, char *argv[]) {

  TIMERSTART(malloc)
  index_t num_entries = 1344;  // number of sequences
  index_t num_features = 1024; // length of all sequences

  /* count total cell updates */
  const value_t CU = num_features * num_features * num_entries * num_entries;
  std::cout << "We are going to process " << CU / 1000000000.0
            << " Giga Cell Updates (GCU)" << std::endl;

  // create host storage and buffers on devices
  value_t *data_cpu = nullptr, // time series on CPU
      *dist_cpu = nullptr,     // distance results on CPU
      *data_gpu,               // time series on GPU
      *dist_gpu;               // distance results on GPU
  hipHostMalloc(&data_cpu,
                 sizeof(value_t) * num_entries * num_features); /* input */
  hipMalloc(&data_gpu, sizeof(value_t) * num_entries * num_features);
  hipHostMalloc(&dist_cpu,
                 sizeof(value_t) * num_entries * num_entries); /* results */
  hipMalloc(&dist_gpu, sizeof(value_t) * num_entries * num_entries);
  CUERR
  TIMERSTOP(malloc)

  /* load data from memory into CPU array, initialize GPU results */
  TIMERSTART(load_data)
  generate_cbf(data_cpu, num_features, num_entries);
  // load_binary(data_cpu, num_features * num_entries,
  //             "../../../data/kernel/dtw_car.bin");
  hipMemcpyAsync(data_gpu, data_cpu,
                  sizeof(value_t) * num_features * num_entries,
                  hipMemcpyHostToDevice);
  CUERR
  hipMemsetAsync(dist_gpu, 0, sizeof(value_t) * num_entries * num_entries);
  CUERR
  TIMERSTOP(load_data)

  /* perform pairwise DTW computation */
  TIMERSTART_CUDA(computation)
  distances(data_gpu, dist_gpu, num_features, num_entries, (float)0.0);
  CUERR
  TIMERSTOP_CUDA(computation)

  /* copy results to cpu */
  TIMERSTART(save_data)
  hipMemcpyAsync(dist_cpu, dist_gpu,
                  sizeof(value_t) * num_entries * num_entries,
                  hipMemcpyDeviceToHost);
  CUERR
  TIMERSTOP(save_data)

#ifdef NV_DEBUG
  /* /1* debug output print *1/ */
  // std::cout << "RESULTS:" << std::endl;
  for (int i = 0; i < num_entries; i++) {
    for (int j = 0; j < num_entries; j++) {
      std::cout << dist_cpu[i * num_entries + j] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
#endif

  TIMERSTART(free)
  hipFree(data_gpu);
  CUERR
  hipFree(dist_gpu);
  CUERR
  hipHostFree(data_cpu);
  CUERR
  hipHostFree(dist_cpu);
  CUERR
  TIMERSTOP(free)

  return 0;
}

#include "hip/hip_runtime.h"
#ifndef MAIN_PROG
#define MAIN_PROG

#include <assert.h>
#include <cstdint>
#include <iostream>
#include <string>

#include "include/DTW.hpp"
#include "include/binary_IO.hpp"
#include "include/cbf_generator.hpp"
#include "include/common.hpp"
#include "include/hpc_helpers.hpp"

using namespace FullDTW;

#ifdef FP16
#include <hip/hip_fp16.h>
#define FLOAT2HALF(a) __float2half2_rn(a)
#define HALF2FLOAT(a) __half2float(a)
typedef __half2 value_ht;
#define FP_PIPES 2
#else
#define FP_PIPES 1
#define FLOAT2HALF(a) a
#define HALF2FLOAT(a) a
typedef float value_ht;
#endif
//------------------time macros--------------------------//
#define TIMERSTART_CUDA(label)                                                 \
  hipSetDevice(0);                                                            \
  hipEvent_t start##label, stop##label;                                       \
  float time##label;                                                           \
  hipEventCreate(&start##label);                                              \
  hipEventCreate(&stop##label);                                               \
  hipEventRecord(start##label, 0);

#define TIMERSTOP_CUDA(label)                                                  \
  hipSetDevice(0);                                                            \
  hipEventRecord(stop##label, 0);                                             \
  hipEventSynchronize(stop##label);                                           \
  hipEventElapsedTime(&time##label, start##label, stop##label);               \
  std::cout << "TIMING: " << time##label << " ms "                             \
            << ((QUERY_LEN) * (REF_LEN)*num_entries * FP_PIPES) /              \
                   (time##label * 1e6)                                         \
            << " GCUPS (" << #label << ")" << std::endl;
//..................time macros............................//

int main(int argc, char *argv[]) {

  index_t num_entries = BLOCK_NUM; // number of sequences

  /* count total cell updates */
  const value_t CU = QUERY_LEN * REF_LEN * num_entries;
  std::cout << "We are going to process " << CU / 1000000000.0
            << " Giga Cell Updates (GCU)" << std::endl;

  // create host storage and buffers on devices
  value_ht *host_query = nullptr,     // time series on CPU
      *host_dist = nullptr,           // distance results on CPU
          *device_query = nullptr,    // time series on GPU
              *device_dist = nullptr, // distance results on GPU
                  *host_ref = nullptr, *device_ref = nullptr;
  raw_t *squiggle_data = nullptr; // random data generated is stored here.
  //------mem allocation---------------//
  TIMERSTART(malloc)
  //--------host mem allocation-----------------//
  hipHostMalloc(&host_query,
                 sizeof(value_ht) * num_entries * QUERY_LEN); /* input */
  hipHostMalloc(&squiggle_data,
                 sizeof(raw_t) * num_entries * QUERY_LEN);    /* input */
  hipHostMalloc(&host_ref, sizeof(value_ht) * REF_LEN);      /* input */
  hipHostMalloc(&host_dist, sizeof(value_ht) * num_entries); /* results */

  //-------dev mem allocation----------//
  hipMalloc(&device_query, sizeof(value_ht) * num_entries * QUERY_LEN);
  hipMalloc(&device_ref, sizeof(value_ht) * REF_LEN);
  hipMalloc(&device_dist, sizeof(value_ht) * num_entries);

  CUERR
  TIMERSTOP(malloc)

  //--------data generation and type conversion-------------------//

  generate_cbf(squiggle_data, QUERY_LEN, num_entries);
#pragma unroll
  for (int i = 0; i < (QUERY_LEN * num_entries); i++) {
    host_query[i] = FLOAT2HALF(squiggle_data[i]);
  }

  /* load data from memory into CPU array, initialize GPU results */
  TIMERSTART(load_data)

  // load_binary(host_dist, QUERY_LEN * num_entries,
  //             "../../../data/kernel/dtw_car.bin");
  hipMemcpyAsync(device_query, host_query,
                  sizeof(value_ht) * QUERY_LEN * num_entries,
                  hipMemcpyHostToDevice);
  hipMemcpyAsync(device_ref,
                  &host_query[0], //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!//
                  sizeof(value_ht) * REF_LEN, hipMemcpyHostToDevice);
  CUERR
  hipMemsetAsync(device_dist, 0, sizeof(value_ht) * num_entries);
  CUERR
  TIMERSTOP(load_data)

  /* perform pairwise DTW computation */
  TIMERSTART_CUDA(computation)
  distances<value_ht, index_t>(device_ref, device_query, device_dist,
                               num_entries, FLOAT2HALF(0));
  CUERR
  TIMERSTOP_CUDA(computation)

  /* copy results to cpu */
  TIMERSTART(save_data)
  hipMemcpyAsync(host_dist, device_dist, sizeof(value_ht) * num_entries,
                  hipMemcpyDeviceToHost);
  CUERR
  TIMERSTOP(save_data)

#ifdef NV_DEBUG
#ifndef FP16
  for (idxt j = 0; j < num_entries; j++) {
    std::cout << HALF2FLOAT(host_dist[j]) << " ";
  }
#else
  for (idxt j = 0; j < num_entries; j++) {
    std::cout << HALF2FLOAT(host_dist[j].x) << " ";
  }
  std::cout << std::endl;
  for (idxt j = 0; j < num_entries; j++) {
    std::cout << HALF2FLOAT(host_dist[j].y) << " ";
  }

#endif
  std::cout << std::endl;
#endif

  TIMERSTART(free)
  hipFree(device_dist);
  CUERR
  hipFree(device_query);
  CUERR
  hipFree(device_ref);
  CUERR
  hipHostFree(host_ref);
  CUERR
  hipHostFree(host_query);
  CUERR
  hipHostFree(host_dist);
  CUERR
  hipHostFree(squiggle_data);
  TIMERSTOP(free)

  return 0;
}

#endif
#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdint>
#include <iostream>
#include <string>

#include "include/DTW.hpp"
#include "include/binary_IO.hpp"
#include "include/cbf_generator.hpp"
#include "include/common.hpp"
#include "include/hpc_helpers.hpp"
#ifdef FP16
#include <hip/hip_fp16.h>
#endif

using namespace FullDTW;

//------------------time macros--------------------------//
#define TIMERSTART_CUDA(label)                                                 \
  hipSetDevice(0);                                                            \
  hipEvent_t start##label, stop##label;                                       \
  float time##label;                                                           \
  hipEventCreate(&start##label);                                              \
  hipEventCreate(&stop##label);                                               \
  hipEventRecord(start##label, 0);

#define TIMERSTOP_CUDA(label)                                                  \
  hipSetDevice(0);                                                            \
  hipEventRecord(stop##label, 0);                                             \
  hipEventSynchronize(stop##label);                                           \
  hipEventElapsedTime(&time##label, start##label, stop##label);               \
  std::cout << "TIMING: " << time##label << " ms "                             \
            << ((QUERY_LEN) * (REF_LEN)*num_entries) / (time##label * 1e6)     \
            << " GCUPS (" << #label << ")" << std::endl;
//..................time macros............................//

#ifndef FP16

int main(int argc, char *argv[]) {

  index_t num_entries = BLOCK_NUM; // number of sequences

  /* count total cell updates */
  const value_t CU = QUERY_LEN * REF_LEN * num_entries;
  std::cout << "We are going to process " << CU / 1000000000.0
            << " Giga Cell Updates (GCU)" << std::endl;

  // create host storage and buffers on devices
  value_t *host_query = nullptr,      // time series on CPU
      *host_dist = nullptr,           // distance results on CPU
          *device_query = nullptr,    // time series on GPU
              *device_dist = nullptr, // distance results on GPU
                  *host_ref = nullptr, *device_ref = nullptr;

  //------mem allocation---------------//
  TIMERSTART(malloc)
  //--------host mem allocation-----------------//
  hipHostMalloc(&host_query,
                 sizeof(value_t) * num_entries * QUERY_LEN); /* input */
  hipHostMalloc(&host_ref, sizeof(value_t) * REF_LEN);      /* input */
  hipHostMalloc(&host_dist, sizeof(value_t) * num_entries); /* results */

  //-------dev mem allocation----------//
  hipMalloc(&device_query, sizeof(value_t) * num_entries * QUERY_LEN);
  hipMalloc(&device_ref, sizeof(value_t) * REF_LEN);
  hipMalloc(&device_dist, sizeof(value_t) * num_entries);

  CUERR
  TIMERSTOP(malloc)

  //--------data generation-------------------//
  generate_cbf(host_query, QUERY_LEN, num_entries);

  /* load data from memory into CPU array, initialize GPU results */
  TIMERSTART(load_data)

  // load_binary(host_dist, QUERY_LEN * num_entries,
  //             "../../../data/kernel/dtw_car.bin");
  hipMemcpyAsync(device_query, host_query,
                  sizeof(value_t) * QUERY_LEN * num_entries,
                  hipMemcpyHostToDevice);
  hipMemcpyAsync(device_ref,
                  &host_query[0], //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!//
                  sizeof(value_t) * REF_LEN, hipMemcpyHostToDevice);
  CUERR
  hipMemsetAsync(device_dist, 0, sizeof(value_t) * num_entries);
  CUERR
  TIMERSTOP(load_data)

  /* perform pairwise DTW computation */
  TIMERSTART_CUDA(computation)
  distances(device_ref, device_query, device_dist, num_entries, (float)0.0);
  CUERR
  TIMERSTOP_CUDA(computation)

  /* copy results to cpu */
  TIMERSTART(save_data)
  hipMemcpyAsync(host_dist, device_dist, sizeof(value_t) * num_entries,
                  hipMemcpyDeviceToHost);
  CUERR
  TIMERSTOP(save_data)

#ifdef NV_DEBUG

  for (idxt j = 0; j < num_entries; j++) {
    std::cout << host_dist[j] << " ";
  }
  std::cout << std::endl;
#endif

  TIMERSTART(free)
  hipFree(device_dist);
  CUERR
  hipFree(device_query);
  CUERR
  hipFree(device_ref);
  CUERR
  hipHostFree(host_ref);
  CUERR
  hipHostFree(host_query);
  CUERR
  hipHostFree(host_dist);
  CUERR
  TIMERSTOP(free)

  return 0;
}

//-------if FP16 is defined -----------------//
#else

int main(int argc, char *argv[]) {

  typedef __half value_ht;
  index_t num_entries = BLOCK_NUM; // number of sequences

  /* count total cell updates */
  const value_t CU = QUERY_LEN * REF_LEN * num_entries;
  std::cout << "We are going to process " << CU / 1000000000.0
            << " Giga Cell Updates (GCU)" << std::endl;

  // create host storage and buffers on devices
  value_ht *host_query = nullptr,     // time series on CPU
      *host_dist = nullptr,           // distance results on CPU
          *device_query = nullptr,    // time series on GPU
              *device_dist = nullptr, // distance results on GPU
                  *host_ref = nullptr, *device_ref = nullptr;
  int *squiggle_data = nullptr; // random data generated is stored here.
  //------mem allocation---------------//
  TIMERSTART(malloc)
  //--------host mem allocation-----------------//
  hipHostMalloc(&host_query,
                 sizeof(value_ht) * num_entries * QUERY_LEN); /* input */
  hipHostMalloc(&squiggle_data,
                 sizeof(int) * num_entries * QUERY_LEN);      /* input */
  hipHostMalloc(&host_ref, sizeof(value_ht) * REF_LEN);      /* input */
  hipHostMalloc(&host_dist, sizeof(value_ht) * num_entries); /* results */

  //-------dev mem allocation----------//
  hipMalloc(&device_query, sizeof(value_ht) * num_entries * QUERY_LEN);
  hipMalloc(&device_ref, sizeof(value_ht) * REF_LEN);
  hipMalloc(&device_dist, sizeof(value_ht) * num_entries);

  CUERR
  TIMERSTOP(malloc)

  //--------data generation and type conversion-------------------//

  generate_cbf(squiggle_data, QUERY_LEN, num_entries);
  for (int i = 0; i < (QUERY_LEN * num_entries); i++)
    host_query[i] = __float2half_rn(squiggle_data[i]);

  /* load data from memory into CPU array, initialize GPU results */
  TIMERSTART(load_data)

  // load_binary(host_dist, QUERY_LEN * num_entries,
  //             "../../../data/kernel/dtw_car.bin");
  hipMemcpyAsync(device_query, host_query,
                  sizeof(value_ht) * QUERY_LEN * num_entries,
                  hipMemcpyHostToDevice);
  hipMemcpyAsync(device_ref,
                  &host_query[0], //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!//
                  sizeof(value_ht) * REF_LEN, hipMemcpyHostToDevice);
  CUERR
  hipMemsetAsync(device_dist, 0, sizeof(value_ht) * num_entries);
  CUERR
  TIMERSTOP(load_data)

  /* perform pairwise DTW computation */
  TIMERSTART_CUDA(computation)
  distances<value_ht, index_t>(device_ref, device_query, device_dist,
                               num_entries, __float2half_rn(0));
  CUERR
  TIMERSTOP_CUDA(computation)

  /* copy results to cpu */
  TIMERSTART(save_data)
  hipMemcpyAsync(host_dist, device_dist, sizeof(value_ht) * num_entries,
                  hipMemcpyDeviceToHost);
  CUERR
  TIMERSTOP(save_data)

#ifdef NV_DEBUG

  for (idxt j = 0; j < num_entries; j++) {
    std::cout << __half2float(host_dist[j]) << " ";
  }
  std::cout << std::endl;
#endif

  TIMERSTART(free)
  hipFree(device_dist);
  CUERR
  hipFree(device_query);
  CUERR
  hipFree(device_ref);
  CUERR
  hipHostFree(host_ref);
  CUERR
  hipHostFree(host_query);
  CUERR
  hipHostFree(host_dist);
  CUERR
  hipHostFree(squiggle_data);
  TIMERSTOP(free)

  return 0;
}

#endif

#include "hip/hip_runtime.h"
#ifndef MAIN_PROG
#define MAIN_PROG

#include <assert.h>
#include <cstdint>
#include <iostream>

#include "include/common.hpp"
#include "include/datatypes.hpp"
#include <stdio.h>
#include <string>
#include <unistd.h>

#include "include/DTW.hpp"
#include "include/binary_IO.hpp"
#include "include/generate_load_squiggle.hpp"
#include "include/hpc_helpers.hpp"
#include "include/load_reference.hpp"
#include "include/normalizer.cu"
#include <unistd.h>

using namespace FullDTW;

//---------------------------------------------------------global
// vars----------------------------------------------------------//
hipStream_t stream_var[STREAM_NUM];

int main(int argc, char **argv) {

  // create host storage and buffers on devices
  value_ht *host_query, // time series on CPU
      *host_dist,       // distance results on CPU
      // *host_ref_coeff1, *host_ref_coeff2,                // re-arranged ref
      // time series on CPU
      *device_query[STREAM_NUM], // time series on GPU
      *device_dist[STREAM_NUM];  // distance results on GPU

  reference_coefficients *h_ref_coeffs,
      *d_ref_coeffs; // struct stores reference genome's coeffs for DTW
  raw_t *raw_array = NULL;

  //****************************************************Target ref loading &
  // re-organization for better mem coalescing & target
  // loading****************************************//

  TIMERSTART(load_target)
  std::string ip_path = argv[1], model_file = argv[2], ref_file = argv[3];

  load_reference *REF_LD = new load_reference;

  REF_LD->ref_loader(ref_file);
  REF_LD->read_kmer_model(model_file);
  ASSERT(hipMallocManaged(&h_ref_coeffs,
                           (sizeof(reference_coefficients) *
                            (REF_LEN)))); // host pinned memory for reference

  REF_LD->load_ref_coeffs(h_ref_coeffs);
  delete REF_LD;

  uint64_t k = 0;

#pragma omp parallel for
  for (index_t l = 0; l < REF_LEN; l += (SEGMENT_SIZE * WARP_SIZE)) {

    for (index_t i = 0; i < SEGMENT_SIZE; i++) {

      for (index_t j = 0; j < WARP_SIZE; j++) {
        h_ref_coeffs[k].coeff1 = h_ref_coeffs[i + (j * SEGMENT_SIZE) +
                                              (l * SEGMENT_SIZE * WARP_SIZE)]
                                     .coeff1;
        h_ref_coeffs[k].coeff2 = h_ref_coeffs[i + (j * SEGMENT_SIZE) +
                                              (l * SEGMENT_SIZE * WARP_SIZE)]
                                     .coeff2;
        // std::cout << HALF2FLOAT(host_ref[k].x) << ",";
        k++;
      }
    }
  }

  ASSERT(
      hipMalloc(&(d_ref_coeffs), (sizeof(reference_coefficients) * REF_LEN)));
  ASSERT(hipMemcpyAsync(
      d_ref_coeffs,
      h_ref_coeffs, //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!//
      (sizeof(reference_coefficients) * REF_LEN), hipMemcpyHostToDevice));

  TIMERSTOP(load_target)

  //*************************************************************LOAD FROM
  // FILE********************************************************//
  TIMERSTART(load_data)
  index_t NUM_READS; // counter to count number of reads to be
                     // processed + reference length
  squiggle_loader *loader = new squiggle_loader;
  loader->load_data(ip_path, raw_array,
                    NUM_READS); // load from input ONT data folder with FAST5
  ASSERT(hipHostMalloc(
      &raw_array,
      (sizeof(raw_t) *
       (NUM_READS * QUERY_LEN)))); // host pinned memory for raw data from FAST5

  loader->load_query(raw_array);

  delete loader;

  //****************************************************NORMALIZER****************************************//
  // normalizer instance - does h2h pinned mem transfer, CUDNN setup andzscore
  // normalization, normalized raw_t output is returned in same array as input
  normalizer *NMZR = new normalizer;
  TIMERSTART(normalizer_kernel)
  NMZR->normalize(raw_array, NUM_READS);
  TIMERSTOP(normalizer_kernel)
  std::cout << "Normalizer processed  " << (QUERY_LEN * NUM_READS)
            << " raw samples in this time\n";

#ifdef NV_DEBUG
  NMZR->print_normalized_query(raw_array, NUM_READS);
#endif

  delete NMZR;
  // normalizartion completed

  //****************************************************FLOAT to
  //__half2****************************************//
  ASSERT(
      hipHostMalloc(&host_query, sizeof(value_ht) * NUM_READS * QUERY_LEN)); /*
                                  input */
  std::cout << "Normalized data:\n";
  for (index_t i = 0; i < NUM_READS; i++) {
    for (index_t j = 0; j < QUERY_LEN; j++) {
      host_query[(i * NUM_READS + j)] =
          FLOAT2HALF(raw_array[(i * NUM_READS + j)]);
    }
  }
  hipHostFree(raw_array);
  TIMERSTOP(load_data)

  //****************************************************MEM
  // allocation****************************************//
  TIMERSTART(malloc)
  //--------------------------------------------------------host mem
  // allocation--------------------------------------------------//

  // ASSERT(hipHostMalloc(&host_ref, sizeof(value_ht) * REF_LEN)); /* input

  ASSERT(hipHostMalloc(&host_dist, sizeof(value_ht) * NUM_READS)); /* results
                                                                     */

  //-------------------------------------------------------------dev mem
  // allocation-------------------------------------------------//

  for (int stream_id = 0; stream_id < STREAM_NUM; stream_id++) {
    ASSERT(hipMalloc(&device_query[stream_id],
                      (sizeof(value_ht) * BLOCK_NUM * QUERY_LEN)));
    ASSERT(hipMalloc(&device_dist[stream_id], sizeof(value_ht) * BLOCK_NUM));
    ASSERT(hipStreamCreate(&stream_var[stream_id]));
  }

  TIMERSTOP(malloc)

  //****************************************************Mem I/O and DTW
  // computation****************************************//
  TIMERSTART_CUDA(concurrent_DTW_kernel_launch)
  //-------------total batches of concurrent workload to & fro
  // device---------------//
  int batch_count = NUM_READS / (BLOCK_NUM * STREAM_NUM);

  if (batch_count > 0) {
    for (int batch_id = 0; batch_id < batch_count; batch_id++) {
      for (int stream_id = 0; stream_id < STREAM_NUM; stream_id++) {
        //----h2d copy-------------//
        ASSERT(hipMemcpyAsync(
            device_query[stream_id],
            &host_query[(batch_id * STREAM_NUM * QUERY_LEN * BLOCK_NUM) +
                        (stream_id * QUERY_LEN * BLOCK_NUM)],
            sizeof(value_ht) * QUERY_LEN * BLOCK_NUM, hipMemcpyHostToDevice,
            stream_var[stream_id]));

        //---------launch kernels------------//
        distances<value_ht, index_t>(d_ref_coeffs, device_query[stream_id],
                                     device_dist[stream_id], BLOCK_NUM,
                                     FLOAT2HALF(0), stream_var[stream_id]);

        //-----d2h copy--------------//
        ASSERT(hipMemcpyAsync(&host_dist[(batch_id * STREAM_NUM * BLOCK_NUM) +
                                          (stream_id * BLOCK_NUM)],
                               device_dist[stream_id],
                               sizeof(value_ht) * BLOCK_NUM,
                               hipMemcpyDeviceToHost, stream_var[stream_id]));
      }
    }
  } else {

    //----h2d copy-------------//
    ASSERT(hipMemcpyAsync(device_query[0], &host_query[0],
                           sizeof(value_ht) * QUERY_LEN * NUM_READS,
                           hipMemcpyHostToDevice, stream_var[0]));

    //---------launch kernels------------//
    distances<value_ht, index_t>(d_ref_coeffs, device_query[0], device_dist[0],
                                 NUM_READS, FLOAT2HALF(0), stream_var[0]);

    //-----d2h copy--------------//
    ASSERT(hipMemcpyAsync(&host_dist[0], device_dist[0],
                           sizeof(value_ht) * NUM_READS, hipMemcpyDeviceToHost,
                           stream_var[0]));
  }
  ASSERT(hipDeviceSynchronize());
  TIMERSTOP_CUDA(concurrent_DTW_kernel_launch)

  /* -----------------------------------------------------------------print
   * output -----------------------------------------------------*/
#ifdef NV_DEBUG
#ifndef FP16
  for (index_t j = 0; j < NUM_READS; j++) {
    std::cout << HALF2FLOAT(host_dist[j]) << " ";
  }
#else
  for (index_t j = 0; j < NUM_READS; j++) {
    std::cout << HALF2FLOAT(host_dist[j].x) << " ";
  }
  std::cout << std::endl;
  for (index_t j = 0; j < NUM_READS; j++) {
    std::cout << HALF2FLOAT(host_dist[j].y) << " ";
  }

#endif
  std::cout << std::endl;
#endif

  /* -----------------------------------------------------------------free
   * memory -----------------------------------------------------*/
  TIMERSTART(free)
  for (int stream_id = 0; stream_id < STREAM_NUM; stream_id++) {
    hipFree(device_dist[stream_id]);
    hipFree(device_query[stream_id]);
  }

  hipHostFree(host_query);
  hipHostFree(host_dist);
  hipFree(h_ref_coeffs);
  hipFree(d_ref_coeffs);
  TIMERSTOP(free)

  return 0;
}

#endif
